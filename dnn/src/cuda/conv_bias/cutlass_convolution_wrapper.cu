#include "hip/hip_runtime.h"
/**
 * \file dnn/src/cuda/conv_bias/cutlass_convolution_wrapper.cu
 * MegEngine is Licensed under the Apache License, Version 2.0 (the "License")
 *
 * Copyright (c) 2014-2021 Megvii Inc. All rights reserved.
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT ARRANTIES OR CONDITIONS OF ANY KIND, either express or
 * implied.
 */
// ignore warning of cutlass
#pragma GCC diagnostic push
#pragma GCC diagnostic ignored "-Wunused-parameter"
#pragma GCC diagnostic ignored "-Wstrict-aliasing"

#if !MEGDNN_TEGRA_X1
#include "cutlass/convolution/device/convolution.h"
#endif
#include "src/common/opr_param_defs_enumv.cuh"
#include "src/cuda/conv_bias/cutlass_convolution_wrapper.cuh"
#pragma GCC diagnostic pop

using namespace megdnn;
using namespace cuda;
using namespace cutlass_wrapper;

/* ====== cutlass kernel wrapper for int8 nchw32 layout ====== */

#if MEGDNN_TEGRA_X1
template <bool NeedLoadFromConstMem>
void megdnn::cuda::cutlass_wrapper::
        do_conv_bias_int8_implicit_gemm_imma_ncdiv32hw32(
                const int8_t* /* d_src */, const int8_t* /* d_filter */,
                const int32_t* /* d_bias */, const int8_t* /* d_z */,
                int8_t* /* d_dst */, int* /* workspace */,
                const convolution::ConvParam& /* param */,
                uint32_t /* nonlinear_mode */, float /* alpha */,
                float /* beta */, float /* gamma */, float /* scale */,
                const GemmCoord& /* threadblock_shape */,
                const GemmCoord& /* warp_shape */, hipStream_t /* stream */) {}
#else
template <bool NeedLoadFromConstMem>
void megdnn::cuda::cutlass_wrapper::
        do_conv_bias_int8_implicit_gemm_imma_ncdiv32hw32(
                const int8_t* d_src, const int8_t* d_filter,
                const int32_t* d_bias, const int8_t* d_z, int8_t* d_dst,
                int* workspace, const convolution::ConvParam& param,
                uint32_t nonlinear_mode, float alpha, float beta, float gamma,
                float scale, const GemmCoord& threadblock_shape,
                const GemmCoord& warp_shape, hipStream_t stream) {
#define DISPATCH_KERNEL_WITH_TILE_SHAPE(threadblock_m_, threadblock_n_,        \
                                        threadblock_k_, warp_m_, warp_n_,      \
                                        warp_k_)                               \
    if (threadblock_shape.m() == threadblock_m_ &&                             \
        threadblock_shape.n() == threadblock_n_ &&                             \
        threadblock_shape.k() == threadblock_k_ &&                             \
        warp_shape.m() == warp_m_ && warp_shape.n() == warp_n_ &&              \
        warp_shape.k() == warp_k_) {                                           \
        using ThreadBlockShape =                                               \
                cutlass::gemm::GemmShape<threadblock_m_, threadblock_n_,       \
                                         threadblock_k_>;                      \
        using WarpShape = cutlass::gemm::GemmShape<warp_m_, warp_n_, warp_k_>; \
        using InstructionShape = cutlass::gemm::GemmShape<8, 8, 16>;           \
        using Convolution = cutlass::conv::device::Convolution<                \
                int8_t, cutlass::layout::TensorNCxHWx<32>, int8_t,             \
                cutlass::layout::TensorCxRSKx<32>, ElementOutput,              \
                cutlass::layout::TensorNCxHWx<32>, int32_t,                    \
                cutlass::layout::TensorNCxHWx<32>, int32_t,                    \
                cutlass::conv::ConvType::kConvolution,                         \
                cutlass::arch::OpClassTensorOp, cutlass::arch::Sm75,           \
                ThreadBlockShape, WarpShape, InstructionShape, EpilogueOp,     \
                cutlass::conv::threadblock::                                   \
                        ConvolutionFpropNCxHWxThreadblockSwizzle,              \
                2, 16, 16, NeedLoadFromConstMem>;                              \
        typename Convolution::ConvolutionParameter conv_param(                 \
                param.n, param.hi, param.wi, param.ci, param.co, param.fh,     \
                param.fw, param.ho, param.wo, param.ph, param.pw, param.sh,    \
                param.sw, 1, 1, cutlass::conv::Mode::kCrossCorrelation);       \
        return cutlass_convolution_wrapper<Convolution>(                       \
                d_src, d_filter, d_bias, d_z, d_dst, workspace, conv_param,    \
                epilogue, stream);                                             \
    }
#define DISPATCH_KERNEL                                                      \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(256, 128, 64, 64, 64, 64);               \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(128, 256, 64, 64, 64, 64);               \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(128, 128, 64, 64, 64, 64);               \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(64, 128, 64, 32, 64, 64);                \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(128, 64, 64, 64, 32, 64);                \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(64, 64, 64, 32, 32, 64);                 \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(32, 64, 64, 32, 16, 64);                 \
    megdnn_assert(false,                                                     \
                  "unsupported threadblock shape (%dx%dx%d) and warp shape " \
                  "(%dx%dx%d)",                                              \
                  threadblock_shape.m(), threadblock_shape.n(),              \
                  threadblock_shape.k(), warp_shape.m(), warp_shape.n(),     \
                  warp_shape.k());
    using ElementOutput = int8_t;
    using ElementAccumulator = int32_t;
    using ElementBias = int32_t;
    using ElementCompute = float;
    using NonlineMode = megdnn::param_enumv::ConvBias::NonlineMode;
    switch (nonlinear_mode) {
        case NonlineMode::IDENTITY: {
            using EpilogueOp =
                    cutlass::epilogue::thread::BiasAddLinearCombinationClamp<
                            ElementOutput, 8, ElementAccumulator, ElementBias,
                            ElementCompute>;
            typename EpilogueOp::Params epilogue{alpha, beta, gamma};
            DISPATCH_KERNEL;
        }
        case NonlineMode::RELU: {
            using EpilogueOp = cutlass::epilogue::thread::
                    BiasAddLinearCombinationReluClamp<
                            ElementOutput, 8, ElementAccumulator, ElementBias,
                            ElementCompute>;
            typename EpilogueOp::Params epilogue{alpha, beta, gamma, 0};
            DISPATCH_KERNEL;
        }
        case NonlineMode::H_SWISH: {
            using EpilogueOp = cutlass::epilogue::thread::
                    BiasAddLinearCombinationHSwishClamp<
                            ElementOutput, 8, ElementAccumulator, ElementBias,
                            ElementCompute>;
            typename EpilogueOp::Params epilogue{alpha, beta, gamma, scale};
            DISPATCH_KERNEL;
        }
        default:
            megdnn_assert(false,
                          "unsupported nonlinear mode for conv bias operator");
    }
#undef DISPATCH_KERNEL_WITH_TILE_SHAPE
#undef DISPATCH_KERNEL
}
#endif

#define INST(need_load_from_const_mem)                                       \
    template void megdnn::cuda::cutlass_wrapper::                            \
            do_conv_bias_int8_implicit_gemm_imma_ncdiv32hw32<                \
                    need_load_from_const_mem>(                               \
                    const int8_t* d_src, const int8_t* d_filter,             \
                    const int32_t* d_bias, const int8_t* d_z, int8_t* d_dst, \
                    int* workspace, const convolution::ConvParam& param,     \
                    uint32_t nonlinear_mode, float alpha, float beta,        \
                    float gamma, float scale,                                \
                    const GemmCoord& threadblock_shape,                      \
                    const GemmCoord& warp_shape, hipStream_t stream);
INST(true);
INST(false);
#undef INST

/* ===== cutlass kernel wrapper for int8 nchw32 layout and nchw4 output ===== */

#if MEGDNN_TEGRA_X1
template <bool NeedLoadFromConstMem>
void megdnn::cuda::cutlass_wrapper::
        do_conv_bias_int8_implicit_gemm_imma_ncdiv32hw32_ncdiv4hw4(
                const int8_t* /* d_src */, const int8_t* /* d_filter */,
                const int32_t* /* d_bias */, const int8_t* /* d_z */,
                int8_t* /* d_dst */, int* /* workspace */,
                const convolution::ConvParam& /* param */,
                uint32_t /* nonlinear_mode */, float /* alpha */,
                float /* beta */, float /* gamma */, float /* scale */,
                const GemmCoord& /* threadblock_shape */,
                const GemmCoord& /* warp_shape */, hipStream_t /* stream */) {}
#else
template <bool NeedLoadFromConstMem>
void megdnn::cuda::cutlass_wrapper::
        do_conv_bias_int8_implicit_gemm_imma_ncdiv32hw32_ncdiv4hw4(
                const int8_t* d_src, const int8_t* d_filter,
                const int32_t* d_bias, const int8_t* d_z, int8_t* d_dst,
                int* workspace, const convolution::ConvParam& param,
                uint32_t nonlinear_mode, float alpha, float beta, float gamma,
                float scale, const GemmCoord& threadblock_shape,
                const GemmCoord& warp_shape, hipStream_t stream) {
#define DISPATCH_KERNEL_WITH_TILE_SHAPE(threadblock_m_, threadblock_n_,        \
                                        threadblock_k_, warp_m_, warp_n_,      \
                                        warp_k_)                               \
    if (threadblock_shape.m() == threadblock_m_ &&                             \
        threadblock_shape.n() == threadblock_n_ &&                             \
        threadblock_shape.k() == threadblock_k_ &&                             \
        warp_shape.m() == warp_m_ && warp_shape.n() == warp_n_ &&              \
        warp_shape.k() == warp_k_) {                                           \
        using ThreadBlockShape =                                               \
                cutlass::gemm::GemmShape<threadblock_m_, threadblock_n_,       \
                                         threadblock_k_>;                      \
        using WarpShape = cutlass::gemm::GemmShape<warp_m_, warp_n_, warp_k_>; \
        using InstructionShape = cutlass::gemm::GemmShape<8, 8, 16>;           \
        using Convolution = cutlass::conv::device::Convolution<                \
                int8_t, cutlass::layout::TensorNCxHWx<32>, int8_t,             \
                cutlass::layout::TensorCxRSKx<32>, ElementOutput,              \
                cutlass::layout::TensorNCxHWx<4>, int32_t,                     \
                cutlass::layout::TensorNCxHWx<4>, int32_t,                     \
                cutlass::conv::ConvType::kConvolution,                         \
                cutlass::arch::OpClassTensorOp, cutlass::arch::Sm75,           \
                ThreadBlockShape, WarpShape, InstructionShape, EpilogueOp,     \
                cutlass::conv::threadblock::                                   \
                        ConvolutionFpropNCxHWxThreadblockSwizzle,              \
                2, 16, 16, NeedLoadFromConstMem>;                              \
        typename Convolution::ConvolutionParameter conv_param(                 \
                param.n, param.hi, param.wi, param.ci, param.co, param.fh,     \
                param.fw, param.ho, param.wo, param.ph, param.pw, param.sh,    \
                param.sw, 1, 1, cutlass::conv::Mode::kCrossCorrelation);       \
        return cutlass_convolution_wrapper<Convolution>(                       \
                d_src, d_filter, d_bias, d_z, d_dst, workspace, conv_param,    \
                epilogue, stream);                                             \
    }
#define DISPATCH_KERNEL                                                      \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(256, 128, 64, 64, 64, 64);               \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(128, 256, 64, 64, 64, 64);               \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(128, 128, 64, 64, 64, 64);               \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(64, 128, 64, 32, 64, 64);                \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(128, 64, 64, 64, 32, 64);                \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(64, 64, 64, 32, 32, 64);                 \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(32, 64, 64, 16, 32, 64);                 \
    megdnn_assert(false,                                                     \
                  "unsupported threadblock shape (%dx%dx%d) and warp shape " \
                  "(%dx%dx%d)",                                              \
                  threadblock_shape.m(), threadblock_shape.n(),              \
                  threadblock_shape.k(), warp_shape.m(), warp_shape.n(),     \
                  warp_shape.k());
    using ElementOutput = int8_t;
    using ElementAccumulator = int32_t;
    using ElementBias = int32_t;
    using ElementCompute = float;
    using NonlineMode = megdnn::param_enumv::ConvBias::NonlineMode;
    switch (nonlinear_mode) {
        case NonlineMode::IDENTITY: {
            using EpilogueOp =
                    cutlass::epilogue::thread::BiasAddLinearCombinationClamp<
                            ElementOutput, 4, ElementAccumulator, ElementBias,
                            ElementCompute>;
            typename EpilogueOp::Params epilogue{alpha, beta, gamma};
            DISPATCH_KERNEL;
        }
        case NonlineMode::RELU: {
            using EpilogueOp = cutlass::epilogue::thread::
                    BiasAddLinearCombinationReluClamp<
                            ElementOutput, 4, ElementAccumulator, ElementBias,
                            ElementCompute>;
            typename EpilogueOp::Params epilogue{alpha, beta, gamma, 0};
            DISPATCH_KERNEL;
        }
        case NonlineMode::H_SWISH: {
            using EpilogueOp = cutlass::epilogue::thread::
                    BiasAddLinearCombinationHSwishClamp<
                            ElementOutput, 4, ElementAccumulator, ElementBias,
                            ElementCompute>;
            typename EpilogueOp::Params epilogue{alpha, beta, gamma, scale};
            DISPATCH_KERNEL;
        }
        default:
            megdnn_assert(false,
                          "unsupported nonlinear mode for conv bias operator");
    }
#undef DISPATCH_KERNEL_WITH_TILE_SHAPE
#undef DISPATCH_KERNEL
}
#endif

#define INST(need_load_from_const_mem)                                       \
    template void megdnn::cuda::cutlass_wrapper::                            \
            do_conv_bias_int8_implicit_gemm_imma_ncdiv32hw32_ncdiv4hw4<      \
                    need_load_from_const_mem>(                               \
                    const int8_t* d_src, const int8_t* d_filter,             \
                    const int32_t* d_bias, const int8_t* d_z, int8_t* d_dst, \
                    int* workspace, const convolution::ConvParam& param,     \
                    uint32_t nonlinear_mode, float alpha, float beta,        \
                    float gamma, float scale,                                \
                    const GemmCoord& threadblock_shape,                      \
                    const GemmCoord& warp_shape, hipStream_t stream);
INST(true);
INST(false);
#undef INST

/* ====== cutlass kernel wrapper for int8 nchw4 layout ====== */

#if MEGDNN_TEGRA_X1
template <bool NeedLoadFromConstMem>
void megdnn::cuda::cutlass_wrapper::
        do_conv_bias_int8_implicit_gemm_dp4a_ncdiv4hw4(
                const int8_t* /* d_src */, const int8_t* /* d_filter */,
                const int32_t* /* d_bias */, const int8_t* /* d_z */,
                int8_t* /* d_dst */, int* /* workspace */,
                const convolution::ConvParam& /* param */,
                uint32_t /* nonlinear_mode */, float /* alpha */,
                float /* beta */, float /* gamma */, float /* scale */,
                const GemmCoord& /* threadblock_shape */,
                const GemmCoord& /* warp_shape */, int /* stages */,
                hipStream_t /* stream */) {}
#else
template <bool NeedLoadFromConstMem>
void megdnn::cuda::cutlass_wrapper::
        do_conv_bias_int8_implicit_gemm_dp4a_ncdiv4hw4(
                const int8_t* d_src, const int8_t* d_filter,
                const int32_t* d_bias, const int8_t* d_z, int8_t* d_dst,
                int* workspace, const convolution::ConvParam& param,
                uint32_t nonlinear_mode, float alpha, float beta, float gamma,
                float scale, const GemmCoord& threadblock_shape,
                const GemmCoord& warp_shape, int stages, hipStream_t stream) {
#define DISPATCH_KERNEL_WITH_TILE_SHAPE(threadblock_m_, threadblock_n_,        \
                                        threadblock_k_, warp_m_, warp_n_,      \
                                        warp_k_, stage_, aligned_)             \
    if (threadblock_shape.m() == threadblock_m_ &&                             \
        threadblock_shape.n() == threadblock_n_ &&                             \
        threadblock_shape.k() == threadblock_k_ &&                             \
        warp_shape.m() == warp_m_ && warp_shape.n() == warp_n_ &&              \
        warp_shape.k() == warp_k_ && stages == stage_) {                       \
        using ThreadBlockShape =                                               \
                cutlass::gemm::GemmShape<threadblock_m_, threadblock_n_,       \
                                         threadblock_k_>;                      \
        using WarpShape = cutlass::gemm::GemmShape<warp_m_, warp_n_, warp_k_>; \
        using InstructionShape = cutlass::gemm::GemmShape<1, 1, 4>;            \
        using Convolution = cutlass::conv::device::Convolution<                \
                int8_t, cutlass::layout::TensorNCxHWx<4>, int8_t,              \
                cutlass::layout::TensorCxRSKx<4>, ElementOutput,               \
                cutlass::layout::TensorNCxHWx<4>, int32_t,                     \
                cutlass::layout::TensorNCxHWx<4>, int32_t,                     \
                cutlass::conv::ConvType::kConvolution,                         \
                cutlass::arch::OpClassSimt, cutlass::arch::Sm61,               \
                ThreadBlockShape, WarpShape, InstructionShape, EpilogueOp,     \
                cutlass::conv::threadblock::                                   \
                        ConvolutionFpropNCxHWxThreadblockSwizzle,              \
                stage_, 4, aligned_, NeedLoadFromConstMem,                     \
                cutlass::arch::OpMultiplyAdd>;                                 \
        typename Convolution::ConvolutionParameter conv_param(                 \
                param.n, param.hi, param.wi, param.ci, param.co, param.fh,     \
                param.fw, param.ho, param.wo, param.ph, param.pw, param.sh,    \
                param.sw, 1, 1, cutlass::conv::Mode::kCrossCorrelation);       \
        return cutlass_convolution_wrapper<Convolution>(                       \
                d_src, d_filter, d_bias, d_z, d_dst, workspace, conv_param,    \
                epilogue, stream);                                             \
    }
#define DISPATCH_KERNEL                                                      \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(128, 128, 32, 64, 32, 32, 2, 16);        \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(128, 64, 32, 64, 32, 32, 2, 16);         \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(64, 128, 32, 64, 32, 32, 2, 16);         \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(128, 32, 32, 64, 32, 32, 2, 16);         \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(32, 128, 32, 32, 64, 32, 2, 16);         \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(64, 64, 32, 64, 32, 32, 2, 16);          \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(32, 64, 32, 32, 64, 32, 2, 16);          \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(64, 32, 32, 64, 32, 32, 2, 16);          \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(32, 32, 32, 32, 32, 32, 2, 16);          \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(16, 128, 16, 16, 128, 16, 1, 8);         \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(16, 64, 8, 16, 64, 8, 2, 4);             \
    megdnn_assert(false,                                                     \
                  "unsupported threadblock shape (%dx%dx%d) and warp shape " \
                  "(%dx%dx%d)",                                              \
                  threadblock_shape.m(), threadblock_shape.n(),              \
                  threadblock_shape.k(), warp_shape.m(), warp_shape.n(),     \
                  warp_shape.k());
    using ElementOutput = int8_t;
    using ElementAccumulator = int32_t;
    using ElementBias = int32_t;
    using ElementCompute = float;
    using NonlineMode = megdnn::param_enumv::ConvBias::NonlineMode;
    switch (nonlinear_mode) {
        case NonlineMode::IDENTITY: {
            using EpilogueOp =
                    cutlass::epilogue::thread::BiasAddLinearCombinationClamp<
                            ElementOutput, 4, ElementAccumulator, ElementBias,
                            ElementCompute>;
            typename EpilogueOp::Params epilogue{alpha, beta, gamma};
            DISPATCH_KERNEL;
        }
        case NonlineMode::RELU: {
            using EpilogueOp = cutlass::epilogue::thread::
                    BiasAddLinearCombinationReluClamp<
                            ElementOutput, 4, ElementAccumulator, ElementBias,
                            ElementCompute>;
            typename EpilogueOp::Params epilogue{alpha, beta, gamma, 0};
            DISPATCH_KERNEL;
        }
        case NonlineMode::H_SWISH: {
            using EpilogueOp = cutlass::epilogue::thread::
                    BiasAddLinearCombinationHSwishClamp<
                            ElementOutput, 4, ElementAccumulator, ElementBias,
                            ElementCompute>;
            typename EpilogueOp::Params epilogue{alpha, beta, gamma, scale};
            DISPATCH_KERNEL;
        }
        default:
            megdnn_assert(false,
                          "unsupported nonlinear mode for conv bias operator");
    }
#undef DISPATCH_KERNEL_WITH_TILE_SHAPE
#undef DISPATCH_KERNEL
}
#endif

#define INST(need_load_from_const_mem)                                       \
    template void megdnn::cuda::cutlass_wrapper::                            \
            do_conv_bias_int8_implicit_gemm_dp4a_ncdiv4hw4<                  \
                    need_load_from_const_mem>(                               \
                    const int8_t* d_src, const int8_t* d_filter,             \
                    const int32_t* d_bias, const int8_t* d_z, int8_t* d_dst, \
                    int* workspace, const convolution::ConvParam& param,     \
                    uint32_t nonlinear_mode, float alpha, float beta,        \
                    float gamma, float scale,                                \
                    const GemmCoord& threadblock_shape,                      \
                    const GemmCoord& warp_shape, int stages,                 \
                    hipStream_t stream);
INST(true);
INST(false);
#undef INST

/* ====== cutlass kernel wrapper for int8 nchw4 layout and nchw output ====== */

#if MEGDNN_TEGRA_X1
template <bool NeedLoadFromConstMem>
void megdnn::cuda::cutlass_wrapper::
        do_conv_bias_int8_implicit_gemm_dp4a_ncdiv4hw4_nchw(
                const int8_t* /* d_src */, const int8_t* /* d_filter */,
                const float* /* d_bias */, const float* /* d_z */,
                float* /* d_dst */, int* /* workspace */,
                const convolution::ConvParam& /* param */,
                uint32_t /* nonlinear_mode */, float /* alpha */,
                float /* beta */, float /* gamma */, float /* scale */,
                const GemmCoord& /* threadblock_shape */,
                const GemmCoord& /* warp_shape */, int /* stages */,
                hipStream_t /* stream */) {}
#else
template <bool NeedLoadFromConstMem>
void megdnn::cuda::cutlass_wrapper::
        do_conv_bias_int8_implicit_gemm_dp4a_ncdiv4hw4_nchw(
                const int8_t* d_src, const int8_t* d_filter,
                const float* d_bias, const float* d_z, float* d_dst,
                int* workspace, const convolution::ConvParam& param,
                uint32_t nonlinear_mode, float alpha, float beta, float gamma,
                float scale, const GemmCoord& threadblock_shape,
                const GemmCoord& warp_shape, int stages, hipStream_t stream) {
#define DISPATCH_KERNEL_WITH_TILE_SHAPE(threadblock_m_, threadblock_n_,        \
                                        threadblock_k_, warp_m_, warp_n_,      \
                                        warp_k_, stages_, aligned_)            \
    if (threadblock_shape.m() == threadblock_m_ &&                             \
        threadblock_shape.n() == threadblock_n_ &&                             \
        threadblock_shape.k() == threadblock_k_ &&                             \
        warp_shape.m() == warp_m_ && warp_shape.n() == warp_n_ &&              \
        warp_shape.k() == warp_k_ && stages == stages_) {                      \
        using ThreadBlockShape =                                               \
                cutlass::gemm::GemmShape<threadblock_m_, threadblock_n_,       \
                                         threadblock_k_>;                      \
        using WarpShape = cutlass::gemm::GemmShape<warp_m_, warp_n_, warp_k_>; \
        using InstructionShape = cutlass::gemm::GemmShape<1, 1, 4>;            \
        using Convolution = cutlass::conv::device::Convolution<                \
                int8_t, cutlass::layout::TensorNCxHWx<4>, int8_t,              \
                cutlass::layout::TensorCxRSKx<4>, ElementOutput,               \
                cutlass::layout::TensorNCHW, float,                            \
                cutlass::layout::TensorNCHW, int32_t,                          \
                cutlass::conv::ConvType::kConvolution,                         \
                cutlass::arch::OpClassSimt, cutlass::arch::Sm61,               \
                ThreadBlockShape, WarpShape, InstructionShape, EpilogueOp,     \
                cutlass::conv::threadblock::                                   \
                        ConvolutionFpropNCxHWxThreadblockSwizzle,              \
                stages_, 4, aligned_, NeedLoadFromConstMem,                    \
                cutlass::arch::OpMultiplyAdd>;                                 \
        typename Convolution::ConvolutionParameter conv_param(                 \
                param.n, param.hi, param.wi, param.ci, param.co, param.fh,     \
                param.fw, param.ho, param.wo, param.ph, param.pw, param.sh,    \
                param.sw, 1, 1, cutlass::conv::Mode::kCrossCorrelation);       \
        return cutlass_convolution_wrapper<Convolution>(                       \
                d_src, d_filter, d_bias, d_z, d_dst, workspace, conv_param,    \
                epilogue, stream);                                             \
    }
#define DISPATCH_KERNEL                                                      \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(128, 128, 32, 64, 32, 32, 2, 16);        \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(128, 64, 32, 64, 32, 32, 2, 16);         \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(64, 128, 32, 64, 32, 32, 2, 16);         \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(128, 32, 32, 64, 32, 32, 2, 16);         \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(32, 128, 32, 32, 64, 32, 2, 16);         \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(64, 64, 32, 64, 32, 32, 2, 16);          \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(32, 64, 32, 32, 64, 32, 2, 16);          \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(64, 32, 32, 64, 32, 32, 2, 16);          \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(32, 32, 32, 32, 32, 32, 2, 16);          \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(16, 128, 16, 16, 128, 16, 1, 8);         \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(16, 64, 8, 16, 64, 8, 2, 4);             \
    megdnn_assert(false,                                                     \
                  "unsupported threadblock shape (%dx%dx%d) and warp shape " \
                  "(%dx%dx%d)",                                              \
                  threadblock_shape.m(), threadblock_shape.n(),              \
                  threadblock_shape.k(), warp_shape.m(), warp_shape.n(),     \
                  warp_shape.k());
    using ElementOutput = float;
    using ElementAccumulator = int32_t;
    using ElementBias = float;
    using ElementCompute = float;
    using NonlineMode = megdnn::param_enumv::ConvBias::NonlineMode;
    switch (nonlinear_mode) {
        case NonlineMode::IDENTITY: {
            using EpilogueOp =
                    cutlass::epilogue::thread::BiasAddLinearCombination<
                            ElementOutput, 1, ElementAccumulator, ElementBias,
                            ElementCompute>;
            typename EpilogueOp::Params epilogue{alpha, beta, gamma};
            DISPATCH_KERNEL;
        }
        case NonlineMode::RELU: {
            using EpilogueOp =
                    cutlass::epilogue::thread::BiasAddLinearCombinationRelu<
                            ElementOutput, 1, ElementAccumulator, ElementBias,
                            ElementCompute>;
            typename EpilogueOp::Params epilogue{alpha, beta, gamma, 0};
            DISPATCH_KERNEL;
        }
        case NonlineMode::H_SWISH: {
            using EpilogueOp =
                    cutlass::epilogue::thread::BiasAddLinearCombinationHSwish<
                            ElementOutput, 1, ElementAccumulator, ElementBias,
                            ElementCompute>;
            typename EpilogueOp::Params epilogue{alpha, beta, gamma, scale};
            DISPATCH_KERNEL;
        }
        default:
            megdnn_assert(false,
                          "unsupported nonlinear mode for conv bias operator");
    }
#undef DISPATCH_KERNEL_WITH_TILE_SHAPE
#undef DISPATCH_KERNEL
}
#endif

#define INST(need_load_from_const_mem)                                   \
    template void megdnn::cuda::cutlass_wrapper::                        \
            do_conv_bias_int8_implicit_gemm_dp4a_ncdiv4hw4_nchw<         \
                    need_load_from_const_mem>(                           \
                    const int8_t* d_src, const int8_t* d_filter,         \
                    const float* d_bias, const float* d_z, float* d_dst, \
                    int* workspace, const convolution::ConvParam& param, \
                    uint32_t nonlinear_mode, float alpha, float beta,    \
                    float gamma, float scale,                            \
                    const GemmCoord& threadblock_shape,                  \
                    const GemmCoord& warp_shape, int stages,             \
                    hipStream_t stream);
INST(true);
INST(false);
#undef INST

/* ===== cutlass kernel wrapper for int8 nchw4 layout and nchw32 output ===== */

#if MEGDNN_TEGRA_X1
template <bool NeedLoadFromConstMem>
void megdnn::cuda::cutlass_wrapper::
        do_conv_bias_int8_implicit_gemm_dp4a_ncdiv4hw4_ncdiv32hw32(
                const int8_t* /* d_src */, const int8_t* /* d_filter */,
                const int32_t* /* d_bias */, const int8_t* /* d_z */,
                int8_t* /* d_dst */, int* /* workspace */,
                const convolution::ConvParam& /* param */,
                uint32_t /* nonlinear_mode */, float /* alpha */,
                float /* beta */, float /* gamma */, float /* scale */,
                const GemmCoord& /* threadblock_shape */,
                const GemmCoord& /* warp_shape */, int /* stages */,
                hipStream_t /* stream */) {}
#else
template <bool NeedLoadFromConstMem>
void megdnn::cuda::cutlass_wrapper::
        do_conv_bias_int8_implicit_gemm_dp4a_ncdiv4hw4_ncdiv32hw32(
                const int8_t* d_src, const int8_t* d_filter,
                const int32_t* d_bias, const int8_t* d_z, int8_t* d_dst,
                int* workspace, const convolution::ConvParam& param,
                uint32_t nonlinear_mode, float alpha, float beta, float gamma,
                float scale, const GemmCoord& threadblock_shape,
                const GemmCoord& warp_shape, int stages, hipStream_t stream) {
#define DISPATCH_KERNEL_WITH_TILE_SHAPE(threadblock_m_, threadblock_n_,        \
                                        threadblock_k_, warp_m_, warp_n_,      \
                                        warp_k_, stages_, aligned_)            \
    if (threadblock_shape.m() == threadblock_m_ &&                             \
        threadblock_shape.n() == threadblock_n_ &&                             \
        threadblock_shape.k() == threadblock_k_ &&                             \
        warp_shape.m() == warp_m_ && warp_shape.n() == warp_n_ &&              \
        warp_shape.k() == warp_k_ && stages == stages_) {                      \
        using ThreadBlockShape =                                               \
                cutlass::gemm::GemmShape<threadblock_m_, threadblock_n_,       \
                                         threadblock_k_>;                      \
        using WarpShape = cutlass::gemm::GemmShape<warp_m_, warp_n_, warp_k_>; \
        using InstructionShape = cutlass::gemm::GemmShape<1, 1, 4>;            \
        using Convolution = cutlass::conv::device::Convolution<                \
                int8_t, cutlass::layout::TensorNCxHWx<4>, int8_t,              \
                cutlass::layout::TensorCxRSKx<4>, ElementOutput,               \
                cutlass::layout::TensorNCxHWx<32>, int32_t,                    \
                cutlass::layout::TensorNCxHWx<32>, int32_t,                    \
                cutlass::conv::ConvType::kConvolution,                         \
                cutlass::arch::OpClassSimt, cutlass::arch::Sm61,               \
                ThreadBlockShape, WarpShape, InstructionShape, EpilogueOp,     \
                cutlass::conv::threadblock::                                   \
                        ConvolutionFpropNCxHWxThreadblockSwizzle,              \
                stages_, 4, aligned_, NeedLoadFromConstMem,                    \
                cutlass::arch::OpMultiplyAdd>;                                 \
        typename Convolution::ConvolutionParameter conv_param(                 \
                param.n, param.hi, param.wi, param.ci, param.co, param.fh,     \
                param.fw, param.ho, param.wo, param.ph, param.pw, param.sh,    \
                param.sw, 1, 1, cutlass::conv::Mode::kCrossCorrelation);       \
        return cutlass_convolution_wrapper<Convolution>(                       \
                d_src, d_filter, d_bias, d_z, d_dst, workspace, conv_param,    \
                epilogue, stream);                                             \
    }
#define DISPATCH_KERNEL                                                      \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(128, 128, 32, 64, 32, 32, 2, 16);        \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(128, 64, 32, 64, 32, 32, 2, 16);         \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(64, 128, 32, 64, 32, 32, 2, 16);         \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(128, 32, 32, 64, 32, 32, 2, 16);         \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(32, 128, 32, 32, 64, 32, 2, 16);         \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(64, 64, 32, 64, 32, 32, 2, 16);          \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(32, 64, 32, 32, 64, 32, 2, 16);          \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(64, 32, 32, 64, 32, 32, 2, 16);          \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(32, 32, 32, 32, 32, 32, 2, 16);          \
    megdnn_assert(false,                                                     \
                  "unsupported threadblock shape (%dx%dx%d) and warp shape " \
                  "(%dx%dx%d)",                                              \
                  threadblock_shape.m(), threadblock_shape.n(),              \
                  threadblock_shape.k(), warp_shape.m(), warp_shape.n(),     \
                  warp_shape.k());
    using ElementOutput = int8_t;
    using ElementAccumulator = int32_t;
    using ElementBias = int32_t;
    using ElementCompute = float;
    using NonlineMode = megdnn::param_enumv::ConvBias::NonlineMode;
    switch (nonlinear_mode) {
        case NonlineMode::IDENTITY: {
            using EpilogueOp =
                    cutlass::epilogue::thread::BiasAddLinearCombinationClamp<
                            ElementOutput, 4, ElementAccumulator, ElementBias,
                            ElementCompute>;
            typename EpilogueOp::Params epilogue{alpha, beta, gamma};
            DISPATCH_KERNEL;
        }
        case NonlineMode::RELU: {
            using EpilogueOp = cutlass::epilogue::thread::
                    BiasAddLinearCombinationReluClamp<
                            ElementOutput, 4, ElementAccumulator, ElementBias,
                            ElementCompute>;
            typename EpilogueOp::Params epilogue{alpha, beta, gamma, 0};
            DISPATCH_KERNEL;
        }
        case NonlineMode::H_SWISH: {
            using EpilogueOp = cutlass::epilogue::thread::
                    BiasAddLinearCombinationHSwishClamp<
                            ElementOutput, 4, ElementAccumulator, ElementBias,
                            ElementCompute>;
            typename EpilogueOp::Params epilogue{alpha, beta, gamma, scale};
            DISPATCH_KERNEL;
        }
        default:
            megdnn_assert(false,
                          "unsupported nonlinear mode for conv bias operator");
    }
#undef DISPATCH_KERNEL_WITH_TILE_SHAPE
#undef DISPATCH_KERNEL
}
#endif

#define INST(need_load_from_const_mem)                                       \
    template void megdnn::cuda::cutlass_wrapper::                            \
            do_conv_bias_int8_implicit_gemm_dp4a_ncdiv4hw4_ncdiv32hw32<      \
                    need_load_from_const_mem>(                               \
                    const int8_t* d_src, const int8_t* d_filter,             \
                    const int32_t* d_bias, const int8_t* d_z, int8_t* d_dst, \
                    int* workspace, const convolution::ConvParam& param,     \
                    uint32_t nonlinear_mode, float alpha, float beta,        \
                    float gamma, float scale,                                \
                    const GemmCoord& threadblock_shape,                      \
                    const GemmCoord& warp_shape, int stages,                 \
                    hipStream_t stream);
INST(true);
INST(false);
#undef INST

/* ====== cutlass kernel wrapper for int4 x int4 nchw64 layout ====== */

#if MEGDNN_TEGRA_X1
template <bool NeedLoadFromConstMem>
void megdnn::cuda::cutlass_wrapper::
        do_conv_bias_int4_int4_implicit_gemm_imma_ncdiv64hw64(
                const int8_t* /* d_src */, const int8_t* /* d_filter */,
                const int32_t* /* d_bias */, const int8_t* /* d_z */,
                int8_t* /* d_dst */, int* /* workspace */,
                const convolution::ConvParam& /* param */,
                uint32_t /* nonlinear_mode */, float /* alpha */,
                float /* beta */, float /* gamma */, float /* scale */,
                const GemmCoord& /* threadblock_shape */,
                const GemmCoord& /* warp_shape */, hipStream_t /* stream */) {}
#else
template <bool NeedLoadFromConstMem>
void megdnn::cuda::cutlass_wrapper::
        do_conv_bias_int4_int4_implicit_gemm_imma_ncdiv64hw64(
                const int8_t* d_src, const int8_t* d_filter,
                const int32_t* d_bias, const int8_t* d_z, int8_t* d_dst,
                int* workspace, const convolution::ConvParam& param,
                uint32_t nonlinear_mode, float alpha, float beta, float gamma,
                float scale, const GemmCoord& threadblock_shape,
                const GemmCoord& warp_shape, hipStream_t stream) {
#define DISPATCH_KERNEL_WITH_TILE_SHAPE(threadblock_m_, threadblock_n_,        \
                                        threadblock_k_, warp_m_, warp_n_,      \
                                        warp_k_)                               \
    if (threadblock_shape.m() == threadblock_m_ &&                             \
        threadblock_shape.n() == threadblock_n_ &&                             \
        threadblock_shape.k() == threadblock_k_ &&                             \
        warp_shape.m() == warp_m_ && warp_shape.n() == warp_n_ &&              \
        warp_shape.k() == warp_k_) {                                           \
        using ThreadBlockShape =                                               \
                cutlass::gemm::GemmShape<threadblock_m_, threadblock_n_,       \
                                         threadblock_k_>;                      \
        using WarpShape = cutlass::gemm::GemmShape<warp_m_, warp_n_, warp_k_>; \
        using InstructionShape = cutlass::gemm::GemmShape<8, 8, 32>;           \
        using Convolution = cutlass::conv::device::Convolution<                \
                cutlass::int4b_t, cutlass::layout::TensorNCxHWx<64>,           \
                cutlass::int4b_t, cutlass::layout::TensorCxRSKx<64>,           \
                ElementOutput, cutlass::layout::TensorNCxHWx<64>, int32_t,     \
                cutlass::layout::TensorNCxHWx<64>, int32_t,                    \
                cutlass::conv::ConvType::kConvolution,                         \
                cutlass::arch::OpClassTensorOp, cutlass::arch::Sm75,           \
                ThreadBlockShape, WarpShape, InstructionShape, EpilogueOp,     \
                cutlass::conv::threadblock::                                   \
                        ConvolutionFpropNCxHWxThreadblockSwizzle,              \
                2, 32, 32, NeedLoadFromConstMem>;                              \
        typename Convolution::ConvolutionParameter conv_param(                 \
                param.n, param.hi, param.wi, param.ci, param.co, param.fh,     \
                param.fw, param.ho, param.wo, param.ph, param.pw, param.sh,    \
                param.sw, 1, 1, cutlass::conv::Mode::kCrossCorrelation);       \
        return cutlass_convolution_wrapper<Convolution>(                       \
                reinterpret_cast<const cutlass::int4b_t*>(d_src),              \
                reinterpret_cast<const cutlass::int4b_t*>(d_filter), d_bias,   \
                reinterpret_cast<const cutlass::int4b_t*>(d_z),                \
                reinterpret_cast<cutlass::int4b_t*>(d_dst), workspace,         \
                conv_param, epilogue, stream);                                 \
    }
#define DISPATCH_KERNEL                                                      \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(128, 128, 128, 64, 64, 128);             \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(256, 128, 128, 64, 64, 128);             \
    megdnn_assert(false,                                                     \
                  "unsupported threadblock shape (%dx%dx%d) and warp shape " \
                  "(%dx%dx%d)",                                              \
                  threadblock_shape.m(), threadblock_shape.n(),              \
                  threadblock_shape.k(), warp_shape.m(), warp_shape.n(),     \
                  warp_shape.k());
    using ElementOutput = cutlass::int4b_t;
    using ElementAccumulator = int32_t;
    using ElementBias = int32_t;
    using ElementCompute = float;
    using NonlineMode = megdnn::param_enumv::ConvBias::NonlineMode;
    switch (nonlinear_mode) {
        case NonlineMode::IDENTITY: {
            using EpilogueOp =
                    cutlass::epilogue::thread::BiasAddLinearCombinationClamp<
                            ElementOutput, 16, ElementAccumulator, ElementBias,
                            ElementCompute>;
            typename EpilogueOp::Params epilogue{alpha, beta, gamma};
            DISPATCH_KERNEL;
        }
        case NonlineMode::RELU: {
            using EpilogueOp = cutlass::epilogue::thread::
                    BiasAddLinearCombinationReluClamp<
                            ElementOutput, 16, ElementAccumulator, ElementBias,
                            ElementCompute>;
            typename EpilogueOp::Params epilogue{alpha, beta, gamma, 0};
            DISPATCH_KERNEL;
        }
        case NonlineMode::H_SWISH: {
            using EpilogueOp = cutlass::epilogue::thread::
                    BiasAddLinearCombinationHSwishClamp<
                            ElementOutput, 16, ElementAccumulator, ElementBias,
                            ElementCompute>;
            typename EpilogueOp::Params epilogue{alpha, beta, gamma, scale};
            DISPATCH_KERNEL;
        }
        default:
            megdnn_assert(false,
                          "unsupported nonlinear mode for conv bias operator");
    }
#undef DISPATCH_KERNEL_WITH_TILE_SHAPE
#undef DISPATCH_KERNEL
}
#endif

#define INST(need_load_from_const_mem)                                       \
    template void megdnn::cuda::cutlass_wrapper::                            \
            do_conv_bias_int4_int4_implicit_gemm_imma_ncdiv64hw64<           \
                    need_load_from_const_mem>(                               \
                    const int8_t* d_src, const int8_t* d_filter,             \
                    const int32_t* d_bias, const int8_t* d_z, int8_t* d_dst, \
                    int* workspace, const convolution::ConvParam& param,     \
                    uint32_t nonlinear_mode, float alpha, float beta,        \
                    float gamma, float scale,                                \
                    const GemmCoord& threadblock_shape,                      \
                    const GemmCoord& warp_shape, hipStream_t stream);
INST(true);
#undef INST

/* ====== cutlass kernel wrapper for uint4 x int4 nchw64 layout ====== */

#if MEGDNN_TEGRA_X1
template <bool NeedLoadFromConstMem>
void megdnn::cuda::cutlass_wrapper::
        do_conv_bias_uint4_int4_implicit_gemm_imma_ncdiv64hw64(
                const uint8_t* /* d_src */, const int8_t* /* d_filter */,
                const int32_t* /* d_bias */, const uint8_t* /* d_z */,
                uint8_t* /* d_dst */, int* /* workspace */,
                const convolution::ConvParam& /* param */,
                uint32_t /* nonlinear_mode */, float /* alpha */,
                float /* beta */, float /* gamma */, float /* delta */,
                float /* theta */, float /* scale */,
                uint8_t /* src_zero_point */,
                const GemmCoord& /* threadblock_shape */,
                const GemmCoord& /* warp_shape */, hipStream_t /* stream */) {}
#else
template <bool NeedLoadFromConstMem>
void megdnn::cuda::cutlass_wrapper::
        do_conv_bias_uint4_int4_implicit_gemm_imma_ncdiv64hw64(
                const uint8_t* d_src, const int8_t* d_filter,
                const int32_t* d_bias, const uint8_t* d_z, uint8_t* d_dst,
                int* workspace, const convolution::ConvParam& param,
                uint32_t nonlinear_mode, float alpha, float beta, float gamma,
                float delta, float theta, float /* scale */,
                uint8_t src_zero_point, const GemmCoord& threadblock_shape,
                const GemmCoord& warp_shape, hipStream_t stream) {
#define DISPATCH_KERNEL_WITH_TILE_SHAPE(threadblock_m_, threadblock_n_,        \
                                        threadblock_k_, warp_m_, warp_n_,      \
                                        warp_k_)                               \
    if (threadblock_shape.m() == threadblock_m_ &&                             \
        threadblock_shape.n() == threadblock_n_ &&                             \
        threadblock_shape.k() == threadblock_k_ &&                             \
        warp_shape.m() == warp_m_ && warp_shape.n() == warp_n_ &&              \
        warp_shape.k() == warp_k_) {                                           \
        using ThreadBlockShape =                                               \
                cutlass::gemm::GemmShape<threadblock_m_, threadblock_n_,       \
                                         threadblock_k_>;                      \
        using WarpShape = cutlass::gemm::GemmShape<warp_m_, warp_n_, warp_k_>; \
        using InstructionShape = cutlass::gemm::GemmShape<8, 8, 32>;           \
        using Convolution = cutlass::conv::device::Convolution<                \
                cutlass::uint4b_t, cutlass::layout::TensorNCxHWx<64>,          \
                cutlass::int4b_t, cutlass::layout::TensorCxRSKx<64>,           \
                ElementOutput, cutlass::layout::TensorNCxHWx<64>, int32_t,     \
                cutlass::layout::TensorNCxHWx<64>, int32_t,                    \
                cutlass::conv::ConvType::kConvolution,                         \
                cutlass::arch::OpClassTensorOp, cutlass::arch::Sm75,           \
                ThreadBlockShape, WarpShape, InstructionShape, EpilogueOp,     \
                cutlass::conv::threadblock::                                   \
                        ConvolutionFpropNCxHWxThreadblockSwizzle,              \
                2, 32, 32, NeedLoadFromConstMem>;                              \
        typename Convolution::ConvolutionParameter conv_param(                 \
                param.n, param.hi, param.wi, param.ci, param.co, param.fh,     \
                param.fw, param.ho, param.wo, param.ph, param.pw, param.sh,    \
                param.sw, 1, 1, cutlass::conv::Mode::kCrossCorrelation);       \
        return cutlass_convolution_wrapper<Convolution>(                       \
                reinterpret_cast<const cutlass::uint4b_t*>(d_src),             \
                reinterpret_cast<const cutlass::int4b_t*>(d_filter), d_bias,   \
                reinterpret_cast<const cutlass::uint4b_t*>(d_z),               \
                reinterpret_cast<cutlass::uint4b_t*>(d_dst), workspace,        \
                conv_param, epilogue, stream, {src_zero_point});               \
    }
#define DISPATCH_KERNEL                                                      \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(128, 128, 128, 64, 64, 128);             \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(256, 128, 128, 64, 64, 128);             \
    megdnn_assert(false,                                                     \
                  "unsupported threadblock shape (%dx%dx%d) and warp shape " \
                  "(%dx%dx%d)",                                              \
                  threadblock_shape.m(), threadblock_shape.n(),              \
                  threadblock_shape.k(), warp_shape.m(), warp_shape.n(),     \
                  warp_shape.k());
    using ElementOutput = cutlass::uint4b_t;
    using ElementAccumulator = int32_t;
    using ElementBias = int32_t;
    using ElementCompute = float;
    using NonlineMode = megdnn::param_enumv::ConvBias::NonlineMode;
    switch (nonlinear_mode) {
        case NonlineMode::IDENTITY: {
            using EpilogueOp =
                    cutlass::epilogue::thread::BiasAddLinearCombinationClamp<
                            ElementOutput, 16, ElementAccumulator, ElementBias,
                            ElementCompute>;
            typename EpilogueOp::Params epilogue{alpha, beta, gamma,
                                                 delta + theta};
            DISPATCH_KERNEL;
        }
        case NonlineMode::RELU: {
            using EpilogueOp = cutlass::epilogue::thread::
                    BiasAddLinearCombinationReluClamp<
                            ElementOutput, 16, ElementAccumulator, ElementBias,
                            ElementCompute>;
            typename EpilogueOp::Params epilogue{alpha, beta,  gamma,
                                                 0,     delta, theta};
            DISPATCH_KERNEL;
        }
        default:
            megdnn_assert(false,
                          "unsupported nonlinear mode for conv bias operator");
    }
#undef DISPATCH_KERNEL_WITH_TILE_SHAPE
#undef DISPATCH_KERNEL
}
#endif

#define INST(need_load_from_const_mem)                                         \
    template void megdnn::cuda::cutlass_wrapper::                              \
            do_conv_bias_uint4_int4_implicit_gemm_imma_ncdiv64hw64<            \
                    need_load_from_const_mem>(                                 \
                    const uint8_t* d_src, const int8_t* d_filter,              \
                    const int32_t* d_bias, const uint8_t* d_z, uint8_t* d_dst, \
                    int* workspace, const convolution::ConvParam& param,       \
                    uint32_t nonlinear_mode, float alpha, float beta,          \
                    float gamma, float delta, float theta, float scale,        \
                    uint8_t src_zero_point,                                    \
                    const GemmCoord& threadblock_shape,                        \
                    const GemmCoord& warp_shape, hipStream_t stream);
INST(true);
#undef INST

/* ===== cutlass kernel wrapper for nchw4 layout and nhwc output ===== */
#if MEGDNN_TEGRA_X1
template <bool signedness>
void megdnn::cuda::cutlass_wrapper::
        do_conv_bias_int8_implicit_gemm_dp4a_ncdiv4hw4_nhwc(
                const int8_t* /* d_src */, const int8_t* /* d_filter */,
                const int32_t* /* d_bias */, const int8_t* /* d_z */,
                int8_t* /* d_dst */, int* /* workspace */,
                const convolution::ConvParam& /* param */,
                uint32_t /* nonlinear_mode */, float /* alpha */,
                float /* beta */, float /* gamma */, float /* delta */,
                float /* theta */, float /* scale */,
                const GemmCoord& /* threadblock_shape */,
                const GemmCoord& /* warp_shape */, int /* stages */,
                hipStream_t /* stream */) {}
#else
template <bool signedness>
void megdnn::cuda::cutlass_wrapper::
        do_conv_bias_int8_implicit_gemm_dp4a_ncdiv4hw4_nhwc(
                const int8_t* d_src, const int8_t* d_filter,
                const int32_t* d_bias, const int8_t* d_z, int8_t* d_dst,
                int* workspace, const convolution::ConvParam& param,
                uint32_t nonlinear_mode, float alpha, float beta, float gamma,
                float delta, float theta, float scale,
                const GemmCoord& threadblock_shape, const GemmCoord& warp_shape,
                int stages, hipStream_t stream) {
#define DISPATCH_KERNEL_WITH_TILE_SHAPE(threadblock_m_, threadblock_n_,        \
                                        threadblock_k_, warp_m_, warp_n_,      \
                                        warp_k_, stages_, aligned_)            \
    if (threadblock_shape.m() == threadblock_m_ &&                             \
        threadblock_shape.n() == threadblock_n_ &&                             \
        threadblock_shape.k() == threadblock_k_ &&                             \
        warp_shape.m() == warp_m_ && warp_shape.n() == warp_n_ &&              \
        warp_shape.k() == warp_k_ && stages == stages_) {                      \
        using ThreadBlockShape =                                               \
                cutlass::gemm::GemmShape<threadblock_m_, threadblock_n_,       \
                                         threadblock_k_>;                      \
        using WarpShape = cutlass::gemm::GemmShape<warp_m_, warp_n_, warp_k_>; \
        using InstructionShape = cutlass::gemm::GemmShape<1, 1, 4>;            \
        using Convolution = cutlass::conv::device::Convolution<                \
                int8_t, cutlass::layout::TensorNCxHWx<4>, int8_t,              \
                cutlass::layout::TensorCxRSKx<4>, ElementOutput,               \
                cutlass::layout::TensorNHWC, int32_t,                          \
                cutlass::layout::TensorNHWC, int32_t,                          \
                cutlass::conv::ConvType::kConvolution,                         \
                cutlass::arch::OpClassSimt, cutlass::arch::Sm75,               \
                ThreadBlockShape, WarpShape, InstructionShape, EpilogueOp,     \
                cutlass::conv::threadblock::                                   \
                        ConvolutionFpropNCxHWxThreadblockSwizzle,              \
                stages_, 4, aligned_, true, cutlass::arch::OpMultiplyAdd>;     \
        typename Convolution::ConvolutionParameter conv_param(                 \
                param.n, param.hi, param.wi, param.ci, param.co, param.fh,     \
                param.fw, param.ho, param.wo, param.ph, param.pw, param.sh,    \
                param.sw, 1, 1, cutlass::conv::Mode::kCrossCorrelation);       \
        return cutlass_convolution_wrapper<Convolution>(                       \
                d_src, d_filter, d_bias,                                       \
                reinterpret_cast<const ElementOutput*>(d_z),                   \
                reinterpret_cast<ElementOutput*>(d_dst), workspace,            \
                conv_param, epilogue, stream);                                 \
    }
#define DISPATCH_KERNEL                                                      \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(128, 128, 32, 64, 32, 32, 2, 16);        \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(128, 64, 32, 64, 32, 32, 2, 16);         \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(64, 128, 32, 64, 32, 32, 2, 16);         \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(128, 32, 32, 64, 32, 32, 2, 16);         \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(32, 128, 32, 32, 64, 32, 2, 16);         \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(64, 64, 32, 64, 32, 32, 2, 16);          \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(32, 64, 32, 32, 64, 32, 2, 16);          \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(64, 32, 32, 64, 32, 32, 2, 16);          \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(32, 32, 32, 32, 32, 32, 2, 16);          \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(16, 128, 16, 16, 128, 16, 1, 8);         \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(16, 64, 8, 16, 64, 8, 2, 4);             \
    megdnn_assert(false,                                                     \
                  "unsupported threadblock shape (%dx%dx%d) and warp shape " \
                  "(%dx%dx%d)",                                              \
                  threadblock_shape.m(), threadblock_shape.n(),              \
                  threadblock_shape.k(), warp_shape.m(), warp_shape.n(),     \
                  warp_shape.k());
    using ElementOutput = cutlass::integer_subbyte<4, signedness>;
    using ElementAccumulator = int32_t;
    using ElementBias = int32_t;
    using ElementCompute = float;
    using NonlineMode = megdnn::param_enumv::ConvBias::NonlineMode;
    switch (nonlinear_mode) {
        case NonlineMode::IDENTITY: {
            using EpilogueOp =
                    cutlass::epilogue::thread::BiasAddLinearCombinationClamp<
                            ElementOutput, 8, ElementAccumulator, ElementBias,
                            ElementCompute>;
            typename EpilogueOp::Params epilogue{alpha, beta, gamma,
                                                 delta + theta};
            DISPATCH_KERNEL;
        }
        case NonlineMode::RELU: {
            using EpilogueOp = cutlass::epilogue::thread::
                    BiasAddLinearCombinationReluClamp<
                            ElementOutput, 8, ElementAccumulator, ElementBias,
                            ElementCompute>;
            typename EpilogueOp::Params epilogue{alpha, beta,  gamma,
                                                 0,     delta, theta};
            DISPATCH_KERNEL;
        }
        case NonlineMode::H_SWISH: {
            using EpilogueOp = cutlass::epilogue::thread::
                    BiasAddLinearCombinationHSwishClamp<
                            ElementOutput, 8, ElementAccumulator, ElementBias,
                            ElementCompute>;
            typename EpilogueOp::Params epilogue{alpha, beta,  gamma,
                                                 scale, delta, theta};
            DISPATCH_KERNEL;
        }
        default:
            megdnn_assert(false,
                          "unsupported nonlinear mode for conv bias operator");
    }
#undef DISPATCH_KERNEL_WITH_TILE_SHAPE
#undef DISPATCH_KERNEL
}
#endif

#define INST(signedness)                                                     \
    template void megdnn::cuda::cutlass_wrapper::                            \
            do_conv_bias_int8_implicit_gemm_dp4a_ncdiv4hw4_nhwc<signedness>( \
                    const int8_t* d_src, const int8_t* d_filter,             \
                    const int32_t* d_bias, const int8_t* d_z, int8_t* d_dst, \
                    int* workspace, const convolution::ConvParam& param,     \
                    uint32_t nonlinear_mode, float alpha, float beta,        \
                    float gamma, float delta, float theta, float scale,      \
                    const GemmCoord& threadblock_shape,                      \
                    const GemmCoord& warp_shape, int stages,                 \
                    hipStream_t stream);
INST(true);
INST(false);
#undef INST

/* ====== cutlass kernel wrapper for int4 x int4 nchw64 layout ====== */

#if MEGDNN_TEGRA_X1
template <bool NeedLoadFromConstMem>
void megdnn::cuda::cutlass_wrapper::
        do_conv_bias_int4_int4_implicit_gemm_imma_nhwc(
                const int8_t* /* d_src */, const int8_t* /* d_filter */,
                const int32_t* /* d_bias */, const int8_t* /* d_z */,
                int8_t* /* d_dst */, int* /* workspace */,
                const convolution::ConvParam& /* param */,
                uint32_t /* nonlinear_mode */, float /* alpha */,
                float /* beta */, float /* gamma */, float /* scale */,
                const GemmCoord& /* threadblock_shape */,
                const GemmCoord& /* warp_shape */,
                const int32_t /* access_size */, hipStream_t /* stream */) {}
#else
template <bool NeedLoadFromConstMem>
void megdnn::cuda::cutlass_wrapper::
        do_conv_bias_int4_int4_implicit_gemm_imma_nhwc(
                const int8_t* d_src, const int8_t* d_filter,
                const int32_t* d_bias, const int8_t* d_z, int8_t* d_dst,
                int* workspace, const convolution::ConvParam& param,
                uint32_t nonlinear_mode, float alpha, float beta, float gamma,
                float scale, const GemmCoord& threadblock_shape,
                const GemmCoord& warp_shape, const int32_t access_size,
                hipStream_t stream) {
#define DISPATCH_KERNEL_WITH_TILE_SHAPE(threadblock_m_, threadblock_n_,        \
                                        threadblock_k_, warp_m_, warp_n_,      \
                                        warp_k_, access_size_)                 \
    if (threadblock_shape.m() == threadblock_m_ &&                             \
        threadblock_shape.n() == threadblock_n_ &&                             \
        threadblock_shape.k() == threadblock_k_ &&                             \
        warp_shape.m() == warp_m_ && warp_shape.n() == warp_n_ &&              \
        warp_shape.k() == warp_k_ && access_size == access_size_) {            \
        using ThreadBlockShape =                                               \
                cutlass::gemm::GemmShape<threadblock_m_, threadblock_n_,       \
                                         threadblock_k_>;                      \
        using WarpShape = cutlass::gemm::GemmShape<warp_m_, warp_n_, warp_k_>; \
        using InstructionShape = cutlass::gemm::GemmShape<8, 8, 32>;           \
        using Convolution = cutlass::conv::device::Convolution<                \
                cutlass::int4b_t, cutlass::layout::TensorNHWC,                 \
                cutlass::int4b_t, cutlass::layout::TensorNCxHWx<access_size_>, \
                ElementOutput, cutlass::layout::TensorNHWC, int32_t,           \
                cutlass::layout::TensorNHWC, int32_t,                          \
                cutlass::conv::ConvType::kConvolution,                         \
                cutlass::arch::OpClassTensorOp, cutlass::arch::Sm75,           \
                ThreadBlockShape, WarpShape, InstructionShape, EpilogueOp,     \
                cutlass::conv::threadblock::                                   \
                        ConvolutionFpropNHWCThreadblockSwizzle,                \
                2, access_size_, access_size_, NeedLoadFromConstMem,           \
                cutlass::arch::OpMultiplyAddSaturate,                          \
                cutlass::conv::ImplicitGemmMode::GEMM_TN>;                     \
        typename Convolution::ConvolutionParameter conv_param(                 \
                param.n, param.hi, param.wi, param.ci, param.co, param.fh,     \
                param.fw, param.ho, param.wo, param.ph, param.pw, param.sh,    \
                param.sw, 1, 1, cutlass::conv::Mode::kCrossCorrelation);       \
        return cutlass_convolution_wrapper<Convolution>(                       \
                reinterpret_cast<const cutlass::int4b_t*>(d_src),              \
                reinterpret_cast<const cutlass::int4b_t*>(d_filter), d_bias,   \
                reinterpret_cast<const cutlass::int4b_t*>(d_z),                \
                reinterpret_cast<cutlass::int4b_t*>(d_dst), workspace,         \
                conv_param, epilogue, stream);                                 \
    }
#define DISPATCH_KERNEL                                                      \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(128, 32, 64, 64, 32, 64, 32);            \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(128, 32, 64, 64, 32, 64, 16);            \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(128, 32, 64, 64, 32, 64, 8);             \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(128, 64, 64, 64, 64, 64, 32);            \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(128, 64, 64, 64, 64, 64, 16);            \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(128, 64, 64, 64, 64, 64, 8);             \
    megdnn_assert(false,                                                     \
                  "unsupported threadblock shape (%dx%dx%d) and warp shape " \
                  "(%dx%dx%d) and access_size (%d)",                         \
                  threadblock_shape.m(), threadblock_shape.n(),              \
                  threadblock_shape.k(), warp_shape.m(), warp_shape.n(),     \
                  warp_shape.k(), access_size);
    using ElementOutput = cutlass::int4b_t;
    using ElementAccumulator = int32_t;
    using ElementBias = int32_t;
    using ElementCompute = float;
    using NonlineMode = megdnn::param_enumv::ConvBias::NonlineMode;
    switch (nonlinear_mode) {
        case NonlineMode::IDENTITY: {
            using EpilogueOp =
                    cutlass::epilogue::thread::BiasAddLinearCombinationClamp<
                            ElementOutput, 8, ElementAccumulator, ElementBias,
                            ElementCompute>;
            typename EpilogueOp::Params epilogue{alpha, beta, gamma};
            DISPATCH_KERNEL;
        }
        case NonlineMode::RELU: {
            using EpilogueOp = cutlass::epilogue::thread::
                    BiasAddLinearCombinationReluClamp<
                            ElementOutput, 8, ElementAccumulator, ElementBias,
                            ElementCompute>;
            typename EpilogueOp::Params epilogue{alpha, beta, gamma, 0};
            DISPATCH_KERNEL;
        }
        case NonlineMode::H_SWISH: {
            using EpilogueOp = cutlass::epilogue::thread::
                    BiasAddLinearCombinationHSwishClamp<
                            ElementOutput, 8, ElementAccumulator, ElementBias,
                            ElementCompute>;
            typename EpilogueOp::Params epilogue{alpha, beta, gamma, scale};
            DISPATCH_KERNEL;
        }
        default:
            megdnn_assert(false,
                          "unsupported nonlinear mode for conv bias operator");
    }
#undef DISPATCH_KERNEL_WITH_TILE_SHAPE
#undef DISPATCH_KERNEL
}
#endif

#define INST(need_load_from_const_mem)                                       \
    template void megdnn::cuda::cutlass_wrapper::                            \
            do_conv_bias_int4_int4_implicit_gemm_imma_nhwc<                  \
                    need_load_from_const_mem>(                               \
                    const int8_t* d_src, const int8_t* d_filter,             \
                    const int32_t* d_bias, const int8_t* d_z, int8_t* d_dst, \
                    int* workspace, const convolution::ConvParam& param,     \
                    uint32_t nonlinear_mode, float alpha, float beta,        \
                    float gamma, float scale,                                \
                    const GemmCoord& threadblock_shape,                      \
                    const GemmCoord& warp_shape, const int32_t access_size,  \
                    hipStream_t stream);
INST(true);
INST(false);
#undef INST

#if MEGDNN_TEGRA_X1
template <bool NeedLoadFromConstMem>
void megdnn::cuda::cutlass_wrapper::
        do_conv_bias_uint4_int4_implicit_gemm_imma_nhwc(
                const uint8_t* /* d_src */, const int8_t* /* d_filter */,
                const int32_t* /* d_bias */, const uint8_t* /* d_z */,
                uint8_t* /* d_dst */, int* /* workspace */,
                const convolution::ConvParam& /* param */,
                uint32_t /* nonlinear_mode */, float /* alpha */,
                float /* beta */, float /* gamma */, float /* delta */,
                float /* theta */, float /* scale */,
                uint8_t /* src_zero_point */,
                const GemmCoord& /* threadblock_shape */,
                const GemmCoord& /* warp_shape */,
                const int32_t /* access_size */, hipStream_t /* stream */) {}
#else
template <bool NeedLoadFromConstMem>
void megdnn::cuda::cutlass_wrapper::
        do_conv_bias_uint4_int4_implicit_gemm_imma_nhwc(
                const uint8_t* d_src, const int8_t* d_filter,
                const int32_t* d_bias, const uint8_t* d_z, uint8_t* d_dst,
                int* workspace, const convolution::ConvParam& param,
                uint32_t nonlinear_mode, float alpha, float beta, float gamma,
                float delta, float theta, float /* scale */,
                uint8_t src_zero_point, const GemmCoord& threadblock_shape,
                const GemmCoord& warp_shape, const int32_t access_size,
                hipStream_t stream) {
#define DISPATCH_KERNEL_WITH_TILE_SHAPE(threadblock_m_, threadblock_n_,        \
                                        threadblock_k_, warp_m_, warp_n_,      \
                                        warp_k_, access_size_)                 \
    if (threadblock_shape.m() == threadblock_m_ &&                             \
        threadblock_shape.n() == threadblock_n_ &&                             \
        threadblock_shape.k() == threadblock_k_ &&                             \
        warp_shape.m() == warp_m_ && warp_shape.n() == warp_n_ &&              \
        warp_shape.k() == warp_k_ && access_size == access_size_) {            \
        using ThreadBlockShape =                                               \
                cutlass::gemm::GemmShape<threadblock_m_, threadblock_n_,       \
                                         threadblock_k_>;                      \
        using WarpShape = cutlass::gemm::GemmShape<warp_m_, warp_n_, warp_k_>; \
        using InstructionShape = cutlass::gemm::GemmShape<8, 8, 32>;           \
        using Convolution = cutlass::conv::device::Convolution<                \
                cutlass::uint4b_t, cutlass::layout::TensorNHWC,                \
                cutlass::int4b_t, cutlass::layout::TensorNCxHWx<access_size_>, \
                ElementOutput, cutlass::layout::TensorNHWC, int32_t,           \
                cutlass::layout::TensorNHWC, int32_t,                          \
                cutlass::conv::ConvType::kConvolution,                         \
                cutlass::arch::OpClassTensorOp, cutlass::arch::Sm75,           \
                ThreadBlockShape, WarpShape, InstructionShape, EpilogueOp,     \
                cutlass::conv::threadblock::                                   \
                        ConvolutionFpropNHWCThreadblockSwizzle,                \
                2, access_size_, access_size_, NeedLoadFromConstMem,           \
                cutlass::arch::OpMultiplyAddSaturate,                          \
                cutlass::conv::ImplicitGemmMode::GEMM_TN>;                     \
        typename Convolution::ConvolutionParameter conv_param(                 \
                param.n, param.hi, param.wi, param.ci, param.co, param.fh,     \
                param.fw, param.ho, param.wo, param.ph, param.pw, param.sh,    \
                param.sw, 1, 1, cutlass::conv::Mode::kCrossCorrelation);       \
        return cutlass_convolution_wrapper<Convolution>(                       \
                reinterpret_cast<const cutlass::uint4b_t*>(d_src),             \
                reinterpret_cast<const cutlass::int4b_t*>(d_filter), d_bias,   \
                reinterpret_cast<const cutlass::uint4b_t*>(d_z),               \
                reinterpret_cast<cutlass::uint4b_t*>(d_dst), workspace,        \
                conv_param, epilogue, stream, {src_zero_point});               \
    }
#define DISPATCH_KERNEL                                                      \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(128, 32, 64, 64, 32, 64, 32);            \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(128, 32, 64, 64, 32, 64, 16);            \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(128, 32, 64, 64, 32, 64, 8);             \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(128, 64, 64, 64, 64, 64, 32);            \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(128, 64, 64, 64, 64, 64, 16);            \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(128, 64, 64, 64, 64, 64, 8);             \
    megdnn_assert(false,                                                     \
                  "unsupported threadblock shape (%dx%dx%d) and warp shape " \
                  "(%dx%dx%d) and access_size (%d)",                         \
                  threadblock_shape.m(), threadblock_shape.n(),              \
                  threadblock_shape.k(), warp_shape.m(), warp_shape.n(),     \
                  warp_shape.k(), access_size);
    using ElementOutput = cutlass::uint4b_t;
    using ElementAccumulator = int32_t;
    using ElementBias = int32_t;
    using ElementCompute = float;
    using NonlineMode = megdnn::param_enumv::ConvBias::NonlineMode;
    switch (nonlinear_mode) {
        case NonlineMode::IDENTITY: {
            using EpilogueOp =
                    cutlass::epilogue::thread::BiasAddLinearCombinationClamp<
                            ElementOutput, 8, ElementAccumulator, ElementBias,
                            ElementCompute>;
            typename EpilogueOp::Params epilogue{alpha, beta, gamma,
                                                 delta + theta};
            DISPATCH_KERNEL;
        }
        case NonlineMode::RELU: {
            using EpilogueOp = cutlass::epilogue::thread::
                    BiasAddLinearCombinationReluClamp<
                            ElementOutput, 8, ElementAccumulator, ElementBias,
                            ElementCompute>;
            typename EpilogueOp::Params epilogue{alpha, beta,  gamma,
                                                 0,     delta, theta};
            DISPATCH_KERNEL;
        }
        default:
            megdnn_assert(false,
                          "unsupported nonlinear mode for conv bias operator");
    }
#undef DISPATCH_KERNEL_WITH_TILE_SHAPE
#undef DISPATCH_KERNEL
}
#endif

#define INST(need_load_from_const_mem)                                         \
    template void megdnn::cuda::cutlass_wrapper::                              \
            do_conv_bias_uint4_int4_implicit_gemm_imma_nhwc<                   \
                    need_load_from_const_mem>(                                 \
                    const uint8_t* d_src, const int8_t* d_filter,              \
                    const int32_t* d_bias, const uint8_t* d_z, uint8_t* d_dst, \
                    int* workspace, const convolution::ConvParam& param,       \
                    uint32_t nonlinear_mode, float alpha, float beta,          \
                    float gamma, float delta, float theta, float scale,        \
                    uint8_t src_zero_point,                                    \
                    const GemmCoord& threadblock_shape,                        \
                    const GemmCoord& warp_shape, const int32_t access_size,    \
                    hipStream_t stream);
INST(true);
INST(false);
#undef INST

// vim: syntax=cuda.doxygen
